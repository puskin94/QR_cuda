#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <strings.h>
#include <hip/hip_runtime.h>

// in this file we have all the prototypes, including 3 mandatory kernels
#include "qr_cuda.h"

int main() {
  int m = 400;
  int n = 300;
  // int m = 1000;
  // int n = 800;
  int threadsN = 512;
  double *A, *R;
  hipEvent_t start, stop;

  // starting the timer

  // allocating the necessary memory
  A = NULL;
  A = (double*)malloc(m * n * sizeof(double));
  bzero(A, m * n);
  R = NULL;
  R = (double*)malloc(n * n * sizeof(double));
  bzero(R, n * n);

  initMatrix(A, n);
  // starting the algorithm
  gram(A, m, n, R, threadsN);

  free(A);
  free(R);

  //stop = clock();
  //printf("Elapsed time %lf [s]\n", (stop-start)/(double)CLOCKS_PER_SEC);
  return 0;
}

void gram(double* A, int m, int n, double *R, int threadsN){
  if (m < n) {
    printf("m must be higher than n");
    return;
  }

  double *ADevice, *RDevice;
  // setting `threadsN` threads per block
  dim3 dimBlock(threadsN, 1, 1);
  // Allocating some space to the device
  // ** -> pointing to a pointer of GPU
  if (hipSuccess != hipMalloc((void **) &ADevice, m * n, sizeof(double))) {
    printf("[!] Error allocating space to the device for the matrix A\n");
    return;
  }
  if (hipSuccess != hipMalloc((void **) &RDevice, n * n, sizeof(double))) {
    printf("[!] Error allocating space to the device for the matrix R\n");
    return;
  }
  // copying the A matrix to the device
  if (hipSuccess != hipMemcpy(ADevice, A, m * n * sizeof(double), hipMemcpyHostToDevice)) {
    printf("[!] Error copying the matrix A to the device\n");
    return;
  }

  for (int i = 0; i < n; ++i) {
    // dimGrid is `n - i`. Every MP uses 1 || > 1 blocks
    xTA <<< n - i, dimBlock >>> (&RDevice[i * n + i], n - i, &ADevice[i], m, n, &ADevice[ii], n);
    scale <<< m, dimBlock >>> (&ADevice[i], m, n, &RDevice[i * n + i]));
    scale <<< n - i, dimBlock >>> (&RDevice[i * n + i]), n - i, 1, &RDevice[i * n + i]);
    r1_update <<< m, dimBlock >>> (&A[i + 1], m, n - i - 2, n, &A[i], n, &R[i]);
  }

}

/**
* Rank 1 update of columns of A
* A     m x n lda
* col   m x 1 ldc
* coeff 1 x n
*/
__global__ void r1_update(double *A, int m, int n, int lda, double *col, int ldc, double *row){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < m) {
    for (int j = 0; j < n; ++j)
      A[index * lda + j] -= row[j] * col[index * ldc];
}

/**
* Mult. for constant s
* d vector
* m number of elements to change
* ld leading dimension (distance from elements)
*
*/
__global__ void scale(double *d, int m, int ld, double *s){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  double *vec;
  vec = sqrt(*s);

  if (index < map)
    d[index * ld] /= *vec;
}

__global__ void xTA (double *y, int k, double*A, int m, int lda, double *x, int ldx){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < k) {
    for (int i = 0; i < m; ++i) {
      sum += x[i * ldx] * A[index + i * lda];
    }
    y[index] = sum;
  }
}

void initMatrix(double *matrix, int n) {
  /* filling the A matrix using this rule
  A(ii, ii) = ii + 1
  ii = 0, N − 1 */
  for (int i = 0; i < n; ++i)
    matrix[i + i * n] = (double)i + 1.0;
}
